
#include "gtest/gtest.h"
#include "glog/logging.h"
#include "../src/sched/DeviceDriver.h"
#include "../src/sched/DeviceDriver_GPU.h"
#include "../src/sched/DeviceMemoryPointer.h"
#include <iostream>
#include <assert.h>
#include <functional>

__host__ __device__ float _f_add_one(float a, void * const arg){
  return a + *((float *) arg);
}
__device__ FUNC_STRANSFORM f_add_one = _f_add_one;

void test_array_equals_constant(float * array, int n_elements, float c){
	const float EPS = 0.01;
	for(int i=0;i<n_elements;i++){
		ASSERT_NEAR(array[i], c, EPS);
	}
}

TEST(DeviceDriverTest, GPU_MEMSET) {
	
	float numbers[1000];

	GPUDriver driver;
	DeviceMemoryPointer_Local_GPURAM p_gpu(0, NULL, sizeof(float)*1000);
	driver.malloc(&p_gpu);
	driver.memset(&p_gpu, 1);

  	hipMemcpy(numbers, p_gpu.ptr, p_gpu.size_in_byte, hipMemcpyDeviceToHost);
	for(int i=0;i<1000;i++){EXPECT_EQ(numbers[i] != 0.0, true);}

	driver.memset(&p_gpu, 0);
  	hipMemcpy(numbers, p_gpu.ptr, p_gpu.size_in_byte, hipMemcpyDeviceToHost);
	test_array_equals_constant(numbers, 1000, 0.0);

	driver.free(&p_gpu);
}

TEST(DeviceDriverTest, GPU_MEMCPY) {
	float numbers[1000];

	GPUDriver driver;
	DeviceMemoryPointer_Local_GPURAM p_gpu(0, NULL, sizeof(float)*1000);
	driver.malloc(&p_gpu);
	DeviceMemoryPointer_Local_GPURAM p_gpu2(0, NULL, sizeof(float)*1000);
	driver.malloc(&p_gpu2);
	driver.memset(&p_gpu, 0);
	driver.memset(&p_gpu2, 1);

	driver.memcpy(&p_gpu2, &p_gpu);
	driver.memset(&p_gpu, 0);
  	hipMemcpy(numbers, p_gpu2.ptr, p_gpu.size_in_byte, hipMemcpyDeviceToHost);
	test_array_equals_constant(numbers, 1000, 0.0);
}


TEST(DeviceDriverTest, GPU_APPLY) {

	float numbers[1000];

	GPUDriver driver;
	DeviceMemoryPointer_Local_GPURAM p_gpu(0, NULL, sizeof(float)*1000);
	driver.malloc(&p_gpu);
	driver.memset(&p_gpu, 0);

	float one = 1.0;
	driver.sapply(&p_gpu, &f_add_one, &one);

  	hipMemcpy(numbers, p_gpu.ptr, p_gpu.size_in_byte, hipMemcpyDeviceToHost);
	test_array_equals_constant(numbers, 1000, 1.0);
}


/*

TEST(DeviceDriverTest, CPU_CONST_BERN) {
	float numbers[10000];
	test_array_set_constant(numbers, 10000, 1000.0);
	DeviceMemoryPointer_Local_RAM p(numbers, sizeof(float)*10000);
	CPUDriver driver;
	driver.sbernoulli_initialize(&p, 0.2);
	float sum = 0.0;
	for(int i=0;i<10000;i++){
		sum += numbers[i];
	}
	ASSERT_NEAR(sum/10000, 0.2, 0.1);
}

TEST(DeviceDriverTest, CPU_CONST_INIT) {
	float numbers[1000];
	test_array_set_constant(numbers, 1000, 1000.0);
	DeviceMemoryPointer_Local_RAM p(numbers, sizeof(float)*1000);
	CPUDriver driver;
	driver.sconstant_initialize(&p, 0.2);
	test_array_equals_constant(numbers, 1000, 0.2);
}

TEST(DeviceDriverTest, CPU_AXPY) {
	float numbers[1000];
	float numbers2[1000];
	test_array_set_constant(numbers, 1000, 1.0);
	test_array_set_constant(numbers2, 1000, 2.0);
	DeviceMemoryPointer_Local_RAM p1(numbers, sizeof(float)*1000);
	DeviceMemoryPointer_Local_RAM p2(numbers2, sizeof(float)*1000);
	CPUDriver driver;
	float alpha = 0.1;
	driver.smath_axpy(alpha, &p1, &p2);
	test_array_set_constant(numbers, 1000, 1.0);
	test_array_set_constant(numbers2, 1000, alpha*1.0+2.0);
}


TEST(DeviceDriverTest, CPU_AXPBY) {
	float numbers[1000];
	float numbers2[1000];
	test_array_set_constant(numbers, 1000, 1.0);
	test_array_set_constant(numbers2, 1000, 2.0);
	DeviceMemoryPointer_Local_RAM p1(numbers, sizeof(float)*1000);
	DeviceMemoryPointer_Local_RAM p2(numbers2, sizeof(float)*1000);
	CPUDriver driver;
	float alpha = 0.1;
	float beta = 0.5;
	driver.smath_axpby(alpha, &p1, beta, &p2);
	test_array_set_constant(numbers, 1000, 1.0);
	test_array_set_constant(numbers2, 1000, alpha*1.0+beta*2.0);
}
*/

//// MOVE THE FOLLOWING TO A SEPERATE TEST


/*


TEST(DeviceDriverTest, GPU_MEMCPY) {
	float numbers[1000];

	GPUDriver driver;
	DeviceMemoryPointer_Local_GPURAM p_gpu(0, NULL, sizeof(float)*1000);
	driver.malloc(&p_gpu);
	DeviceMemoryPointer_Local_GPURAM p_gpu2(0, NULL, sizeof(float)*1000);
	driver.malloc(&p_gpu2);
	driver.memset(p_gpu, 0);
	driver.memset(p_gpu2, 1);

	driver.memcpy(p_gpu2, p_gpu);
	clEnqueueReadBuffer (driver.queue, (cl_mem) p_gpu2.ptr, CL_TRUE, 0,
		sizeof (float) * 1000, numbers, 0, nullptr, nullptr);
	for(int i=0;i<1000;i++){
		EXPECT_EQ(numbers[i], 0.0);
	}
}

TEST(DeviceDriverTest, GPU_APPLY) {
	float numbers[1000];
	test_array_set_constant(numbers, 1000, 1000.0);
	DeviceMemoryPointer_Local_RAM p(numbers, sizeof(float)*1000);
	GPUDriver driver;
	auto f_set_to_zero = [](float & b) { b = 0; };
	driver.sapply(p, f_set_to_zero);
	test_array_equals_constant(numbers, 1000, 0.0);
}
*/


