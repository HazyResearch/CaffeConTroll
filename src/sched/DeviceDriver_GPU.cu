#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "DeviceDriver.h"
#include "DeviceDriver_GPU.h"

#include "../kernels/include.hxx"


__host__ __device__ float __sconstant_initialize_helper(float a, void * arg){
  return *((float*)arg);
}

template<FUNC_STRANSFORM func>
__global__ void _sapply(float * dst, int numElements, void * const func_curry){
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i < numElements){
    dst[i] = func(dst[i], func_curry);
  }
}

template<FUNC_SREDUCE func>
__global__ void _sreduce(float * dst, int numElements, float * src1, float * src2, 
	void * const func_curry){
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i < numElements){
    dst[i] = func(src1[i], src2[i], func_curry);
  }
}

template<FUNC_IDX_MAPPING idx_func, FUNC_MM_MAPPING func>
__global__ void _spmap(float * dst, float * src, int numElements, int srcSkip,
  void * const idx_func_curry, void * const func_curry){

  int i = (blockDim.x * blockIdx.x + threadIdx.x);
  i = i * srcSkip;
  int src_idx, dst_idx;

  src_idx = i;
  if(src_idx < numElements*srcSkip){
    dst_idx = idx_func(src_idx, idx_func_curry);
    func(&dst[dst_idx/sizeof(float)], &src[src_idx/sizeof(float)], func_curry);
  }
}

template<FPMAP_ID f_id, FPMAP_DATA_READC f_data>
__global__ void _spmap_readc(float* dst, float * src, PMapHelper args){
	const size_t block_x = blockIdx.x;
	const size_t block_y = blockIdx.y;

	//const size_t nRblock = args.sR/args.sBR;
	const size_t nCblock = args.sC/args.sBC;

	Block2D input_block;
	input_block.r = block_x / nCblock;
	input_block.c = block_x % nCblock;
	input_block.d = block_y % args.sD;
	input_block.b = block_y / args.sD;
	input_block.dr = args.sR;
	input_block.dc = args.sC;

	Block2D output_block;
	f_id(&output_block, &input_block, &args);

	const size_t datar = threadIdx.y + input_block.r * args.sR;
	const size_t datac = threadIdx.x + input_block.c * args.sC;

	PointIn2DBlock point;
	point.block = input_block;
	point.data = src[
		args.sR * args.sC * (args.sD * input_block.b + input_block.d) +
		datar + args.sC +
		datac
	];
	point.r = threadIdx.y;
	point.c = threadIdx.x;

	f_data(dst, &output_block, &point, &args);

}


template<FPMAP_ID f_id, FPMAP_DATA_READC f_data>
void GPUDriver::pmap2d_read_coalesce(DeviceMemoryPointer * dst, DeviceMemoryPointer * src, 
    const struct PMapHelper args){

	// input block sizes
	size_t sBR = args.sBR, sBC = args.sBC;
	dim3 threadsPerBlock(sBC, sBR);	// trivial impl -- each input pixel is a single thread
	dim3 numBlocks(args.sR*args.sC/sBC/sBR, args.sD*args.sB);

	_spmap_readc<f_id,f_data><<<numBlocks, threadsPerBlock>>>((float*) dst->ptr, (float*) src->ptr, args);
	err = hipGetLastError();
	if(err != hipSuccess){
	  std::cout << "Fail to launch _spmap_readc"  << "  ERROR " << err << std::endl;
	  assert(false);
	}
	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess){
	  std::cout << "Fail to hipDeviceSynchronize _spmap_readc"  << "  ERROR " << err << std::endl;
	  assert(false);
	}

}


GPUDriver::GPUDriver(){
    hipblasCreate(&handle);
}

DeviceMemoryPointer * GPUDriver::get_device_pointer(void * ptr, size_t size_in_byte){
	// TODO: This has memory leak! Refactor it!
	return new DeviceMemoryPointer_Local_GPURAM(gpu_id, ptr, size_in_byte);
}

void GPUDriver::malloc(DeviceMemoryPointer * dst){
	hipMalloc((void**)&dst->ptr, dst->size_in_byte);
}

void GPUDriver::free(DeviceMemoryPointer * dst){
	hipFree(dst->ptr);
}

void GPUDriver::memcpy(DeviceMemoryPointer * dst, DeviceMemoryPointer * src){
	#ifdef _DO_ASSERT
	assert(dst->size_in_byte == src->size_in_byte);
	#endif
	if(src->type == DEVICEMEMORY_LOCAL_RAM){
  		hipMemcpy(dst->ptr, src->ptr, dst->size_in_byte, hipMemcpyHostToDevice);
	}else if(dst->type == DEVICEMEMORY_LOCAL_RAM){
  		hipMemcpy(dst->ptr, src->ptr, dst->size_in_byte, hipMemcpyDeviceToHost);
	}else{
		hipMemcpy(dst->ptr, src->ptr, dst->size_in_byte, hipMemcpyDeviceToDevice);
	}
}

void GPUDriver::memset(DeviceMemoryPointer * dst, const char value){
	#ifdef _DO_ASSERT
	assert(dst->type==DEVICEMEMORY_LOCAL_RAM);
	#endif
	hipMemset(dst->ptr, value, dst->size_in_byte);
}

template<FUNC_IDX_MAPPING f_dst_pos, FUNC_MM_MAPPING func>
void GPUDriver::parallel_map(DeviceMemoryPointer * dst, DeviceMemoryPointer * src, 
size_t src_skip, DeviceMemoryPointer * const f_dst_pos_curry, DeviceMemoryPointer * const func_curry){

	// create a device version of func_curry
	void * d_func_curry;
	hipMalloc((void**)&d_func_curry, func_curry->size_in_byte);
	hipMemcpy(d_func_curry, func_curry->ptr, func_curry->size_in_byte, hipMemcpyHostToDevice);

	void * d_idx_func_curry;
	hipMalloc((void**)&d_idx_func_curry, f_dst_pos_curry->size_in_byte);
	hipMemcpy(d_idx_func_curry, f_dst_pos_curry->ptr, f_dst_pos_curry->size_in_byte, hipMemcpyHostToDevice);

	// Run.
	const int n_elements =  dst->size_in_byte / src_skip;
	int blocksPerGrid = (n_elements + 1 + threadsPerBlock - 1) / threadsPerBlock;
	_spmap<f_dst_pos,func><<<blocksPerGrid, threadsPerBlock>>>((float*) dst->ptr, (float *) src->ptr,
	  n_elements, src_skip, d_idx_func_curry, d_func_curry);
	err = hipGetLastError();
	if(err != hipSuccess){
	  std::cout << "Fail to launch _spmap"  << "  ERROR " << err << std::endl;
	  assert(false);
	}
	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess){
	  std::cout << "Fail to sync _spmap"  << "  ERROR " << err << std::endl;
	  assert(false);
	}

	hipFree(d_func_curry);
	hipFree(d_idx_func_curry);

}

void GPUDriver::smath_axpy(const float alpha, DeviceMemoryPointer * X, DeviceMemoryPointer * Y)  { 
#ifdef _DO_ASSERT
	assert(X->type==DEVICEMEMORY_LOCAL_RAM);
	assert(Y->type==DEVICEMEMORY_LOCAL_RAM);
	assert(X->size_in_byte==Y->size_in_byte);
#endif
  int n_elements = X->size_in_byte / sizeof(float);
  status = hipblasSaxpy(handle, n_elements, &alpha, (float*)X->ptr, 1, (float*)Y->ptr, 1);
  assert(status == HIPBLAS_STATUS_SUCCESS);
}

template<FUNC_STRANSFORM func>
void GPUDriver::sapply(DeviceMemoryPointer * dst, DeviceMemoryPointer * const func_curry){
	#ifdef _DO_ASSERT
	assert(dst->type==DEVICEMEMORY_LOCAL_RAM);
	assert(dst->size_in_byte % sizeof(float) == 0);
	#endif
	// TODO: Refactoring

	// Second, create a device version of func_curry
	void * d_func_curry;
	hipMalloc((void**)&d_func_curry, func_curry->size_in_byte);
	hipMemcpy(d_func_curry, func_curry->ptr, func_curry->size_in_byte, hipMemcpyHostToDevice);

	// Run.
	const int n_elements =  dst->size_in_byte / sizeof(float);
	int blocksPerGrid = (n_elements + threadsPerBlock - 1) / threadsPerBlock;
	_sapply<func><<<blocksPerGrid, threadsPerBlock>>>((float*) dst->ptr, n_elements, d_func_curry);
	err = hipGetLastError();
	if(err != hipSuccess){
	  std::cout << "Fail to launch _sapply" << "  ERROR " << err << std::endl;
	  assert(false);
	}
	hipDeviceSynchronize();
	err = hipGetLastError();
	assert(err == hipSuccess);

	hipFree(d_func_curry);
}

void GPUDriver::smath_axpby(const float alpha, DeviceMemoryPointer * X, const float beta, DeviceMemoryPointer * Y) { 
#ifdef _DO_ASSERT
  assert(X->size_in_byte == Y->size_in_byte);
  assert(X->size_in_byte % sizeof(float) == 0);
#endif

  int n_elements = X->size_in_byte / sizeof(float);
  status = hipblasSscal(handle, n_elements, &beta, (float*)Y->ptr, 1);
  assert(status == HIPBLAS_STATUS_SUCCESS);

  status = hipblasSaxpy(handle, n_elements, &alpha, (float*)X->ptr, 1, (float*)Y->ptr, 1);
  assert(status == HIPBLAS_STATUS_SUCCESS);

}

void GPUDriver::set_num_threads(const int nThreads) { 
}


void GPUDriver::sgemm(const enum CBLAS_ORDER order, CBLAS_TRANSPOSE TA, CBLAS_TRANSPOSE TB, 
    int M, int N, int K, float alpha, float * pA, int LDA, float * pB, int LDB,
    float beta, float * pC, int LDC){
  
	if(TA == CblasNoTrans && TB == CblasNoTrans){

		hipblasOperation_t ta = HIPBLAS_OP_N;
		hipblasOperation_t tb = HIPBLAS_OP_N;

		status = hipblasSgemm(handle, tb, ta, N, M, K, &alpha, 
			pB, N, pA, K, &beta, pC, N); 

		hipDeviceSynchronize();
		err = hipGetLastError();
		assert(err == hipSuccess);

		assert(status == HIPBLAS_STATUS_SUCCESS);

	}else{
		assert(false);
	}

}

template<FUNC_SREDUCE func>
void GPUDriver::selementwise_reduce2(DeviceMemoryPointer * dst, DeviceMemoryPointer * src1, 
DeviceMemoryPointer * src2, DeviceMemoryPointer * const func_curry){ 

	#ifdef _DO_ASSERT
	assert(dst->size_in_byte == src1->size_in_byte);
	assert(dst->size_in_byte == src2->size_in_byte);
	assert(dst->size_in_byte % sizeof(float) == 0);
	#endif

	// create a device version of func_curry
	void * d_func_curry;
	hipMalloc((void**)&d_func_curry, func_curry->size_in_byte);
	hipMemcpy(d_func_curry, func_curry->ptr, func_curry->size_in_byte, hipMemcpyHostToDevice);

	// Run.
	const int n_elements =  dst->size_in_byte / sizeof(float);
	int blocksPerGrid = (n_elements + threadsPerBlock - 1) / threadsPerBlock;
	_sreduce<func><<<blocksPerGrid, threadsPerBlock>>>((float*) dst->ptr, n_elements, 
	  (float*) src1->ptr, (float*) src2->ptr, d_func_curry);
	err = hipGetLastError();
	if(err != hipSuccess){
	  std::cout << "Fail to launch _sreduce" << std::endl;
	  assert(false);
	}
	hipDeviceSynchronize();
	err = hipGetLastError();
	assert(err == hipSuccess);


}

/**
* This function is called only once. So its speed does not matter.
* TODO: Wrap this up with CURAND.
**/
void GPUDriver::sinitialize_xavier(DeviceMemoryPointer *arr, const size_t n_batch) {
	const size_t n_arr_elements = arr->size_in_byte / sizeof(float);
	const size_t fan_in = n_arr_elements / n_batch;
	const float scale = sqrt(3.0 / fan_in);

	mt19937 gen(rd());
	uniform_real_distribution<float> uni(-scale, scale);
	float * temp = new float[n_arr_elements];
	for(int i=0;i<n_arr_elements;i++){
	  temp[i] = uni(gen);
	}
	hipMemcpy(arr->ptr, temp, arr->size_in_byte, hipMemcpyHostToDevice);
	delete[] temp;
	}

/**
* This function is called only once. So its speed does not matter.
* TODO: Wrap this up with CURAND.
**/
void GPUDriver::sbernoulli_initialize(DeviceMemoryPointer *arr, const float p) {
const size_t n_arr_elements = arr->size_in_byte / sizeof(float);

	mt19937 gen(rd());
	bernoulli_distribution bern(p);
	float * temp = new float[n_arr_elements];
	for(int i=0;i<n_arr_elements;i++){
	  temp[i] = bern(gen);
	}
	hipMemcpy(arr->ptr, temp, arr->size_in_byte, hipMemcpyHostToDevice);
	delete[] temp;

}

/**
* This function is called only once. So its speed does not matter.
* TODO: Wrap this up with CURAND.
**/
void GPUDriver::sgaussian_initialize(DeviceMemoryPointer *arr, const float mean, const float std_dev) {
const size_t n_arr_elements = arr->size_in_byte / sizeof(float);
	mt19937 gen(rd());
	normal_distribution<float> gaussian(mean, std_dev);
	float * temp = new float[n_arr_elements];
	for(int i=0;i<n_arr_elements;i++){
	  temp[i] = gaussian(gen);
	}
	hipMemcpy(arr->ptr, temp, arr->size_in_byte, hipMemcpyHostToDevice);
	delete[] temp;

}

void GPUDriver::sconstant_initialize(DeviceMemoryPointer *arr, const float value){
    DeviceMemoryPointer_Local_RAM pvalue((void*)&value, sizeof(float));
    sapply<__sconstant_initialize_helper>(arr, &pvalue);
}

void * GPUDriver::choose_ptr(void * host, void * device){
	return device;
}

/**
 * This is necessary for template to be instantiated.
 */
template void GPUDriver::pmap2d_read_coalesce<_fpmap_id,_fmap_lower>(DeviceMemoryPointer * dst, 
	DeviceMemoryPointer * src, const struct PMapHelper args);

template void GPUDriver::pmap2d_read_coalesce<_fpmap_id,_fmap_remap>(DeviceMemoryPointer * dst, 
	DeviceMemoryPointer * src, const struct PMapHelper args);

/** All template instantiations for parallel_map **/
template void GPUDriver::parallel_map<_f_idx_strid4_copy,_f_strid4_copy>(DeviceMemoryPointer * dst,
    DeviceMemoryPointer * src, size_t src_skip, DeviceMemoryPointer * const f_dst_pos_curry,
    DeviceMemoryPointer * const func_curry);
// Conv/FC Bias forward
template void GPUDriver::parallel_map<_f_src_to_dst_bias_forward,_f_bias_forward>(DeviceMemoryPointer * dst,
    DeviceMemoryPointer * src, size_t src_skip, DeviceMemoryPointer * const f_dst_pos_curry,
    DeviceMemoryPointer * const func_curry);
// ReLU forward
template void GPUDriver::parallel_map<_f_src_to_dst_relu_forward,_f_relu_forward>(DeviceMemoryPointer * dst,
    DeviceMemoryPointer * src, size_t src_skip, DeviceMemoryPointer * const f_dst_pos_curry,
    DeviceMemoryPointer * const func_curry);
// ReLU backward
template void GPUDriver::parallel_map<_f_src_to_dst_relu_backward,_f_relu_backward>(DeviceMemoryPointer * dst,
    DeviceMemoryPointer * src, size_t src_skip, DeviceMemoryPointer * const f_dst_pos_curry,
    DeviceMemoryPointer * const func_curry);
// Dropout forward train
template void GPUDriver::parallel_map<_f_src_to_dst_dropout_forward,_f_dropout_forward_train>(DeviceMemoryPointer * dst,
    DeviceMemoryPointer * src, size_t src_skip, DeviceMemoryPointer * const f_dst_pos_curry,
    DeviceMemoryPointer * const func_curry);
// Dropout forward test
template void GPUDriver::parallel_map<_f_src_to_dst_dropout_forward,_f_dropout_forward_test>(DeviceMemoryPointer * dst,
    DeviceMemoryPointer * src, size_t src_skip, DeviceMemoryPointer * const f_dst_pos_curry,
    DeviceMemoryPointer * const func_curry);
// Pool forward
template void GPUDriver::parallel_map<_f_src_to_dst_pool_forward,_f_pool_forward>(DeviceMemoryPointer * dst,
    DeviceMemoryPointer * src, size_t src_skip, DeviceMemoryPointer * const f_dst_pos_curry,
    DeviceMemoryPointer * const func_curry);
// Pool backward
template void GPUDriver::parallel_map<_f_src_to_dst_pool_backward,_f_pool_backward>(DeviceMemoryPointer * dst,
    DeviceMemoryPointer * src, size_t src_skip, DeviceMemoryPointer * const f_dst_pos_curry,
    DeviceMemoryPointer * const func_curry);
// LRN forward
template void GPUDriver::parallel_map<_f_src_to_dst_lrn_forward,_f_lrn_forward>(DeviceMemoryPointer * dst,
    DeviceMemoryPointer * src, size_t src_skip, DeviceMemoryPointer * const f_dst_pos_curry,
    DeviceMemoryPointer * const func_curry);
// LRN forward normalize
template void GPUDriver::parallel_map<_f_src_to_dst_lrn_forward,_f_lrn_forward_normalize>(DeviceMemoryPointer * dst,
    DeviceMemoryPointer * src, size_t src_skip, DeviceMemoryPointer * const f_dst_pos_curry,
    DeviceMemoryPointer * const func_curry);
// LRN backward
template void GPUDriver::parallel_map<_f_src_to_dst_lrn_backward,_f_lrn_backward>(DeviceMemoryPointer * dst,
    DeviceMemoryPointer * src, size_t src_skip, DeviceMemoryPointer * const f_dst_pos_curry,
    DeviceMemoryPointer * const func_curry);
// Softmax forward
template void GPUDriver::parallel_map<_f_src_to_dst_softmax_forward,_f_softmax_forward>(DeviceMemoryPointer * dst,
    DeviceMemoryPointer * src, size_t src_skip, DeviceMemoryPointer * const f_dst_pos_curry,
    DeviceMemoryPointer * const func_curry);
// Softmax backward
template void GPUDriver::parallel_map<_f_src_to_dst_softmax_backward,_f_softmax_backward>(DeviceMemoryPointer * dst,
    DeviceMemoryPointer * src, size_t src_skip, DeviceMemoryPointer * const f_dst_pos_curry,
    DeviceMemoryPointer * const func_curry);

template void GPUDriver::sapply<_f_add_one>(DeviceMemoryPointer * dst, DeviceMemoryPointer * const func_curry);

template void GPUDriver::sapply<_f_set>(DeviceMemoryPointer * dst, DeviceMemoryPointer * const func_curry);

template void GPUDriver::selementwise_reduce2<_f_reduce>(DeviceMemoryPointer * dst, 
	DeviceMemoryPointer * src1, DeviceMemoryPointer * src2, DeviceMemoryPointer * const func_curry);

template void GPUDriver::selementwise_reduce2<_f_reduce_mul>(DeviceMemoryPointer * dst, 
	DeviceMemoryPointer * src1, DeviceMemoryPointer * src2, DeviceMemoryPointer * const func_curry);

template void GPUDriver::selementwise_reduce2<_f_reduce_tanhgrad>(DeviceMemoryPointer * dst, 
	DeviceMemoryPointer * src1, DeviceMemoryPointer * src2, DeviceMemoryPointer * const func_curry);

