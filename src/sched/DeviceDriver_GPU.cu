#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "DeviceDriver.h"
#include "DeviceDriver_GPU.h"

#include "../kernels/include.hxx"


__host__ __device__ float __sconstant_initialize_helper(float a, void * arg){
  return *((float*)arg);
}

template<FUNC_STRANSFORM func>
__global__ void _sapply(float * dst, int numElements, void * const func_curry){
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i < numElements){
    dst[i] = func(dst[i], func_curry);
  }
}

template<FUNC_SREDUCE func>
__global__ void _sreduce(float * dst, int numElements, float * src1, float * src2, 
	void * const func_curry){
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i < numElements){
    dst[i] = func(src1[i], src2[i], func_curry);
  }
}

// NOTE: Ensure there are no race conditions when calling this function
// See backward bias calculation for conv and fullyconnected - need to call
// parallel_map multiple times because otherwise output indices overlap
template<FUNC_IDX_MAPPING idx_func, FUNC_MM_MAPPING func>
__global__ void _spmap(float * dst, float * src, int numElements, int srcSkip,
  void * const idx_func_curry, void * const func_curry){
  char * p_dst = (char*) dst;
  char * p_src = (char*) src;
  const size_t src_size = numElements*srcSkip;
  size_t i = (blockDim.x * blockIdx.x + threadIdx.x) * srcSkip;
  if(i < src_size){
    func(&p_dst[idx_func(i, idx_func_curry)], &p_src[i], func_curry, idx_func(i, idx_func_curry));
  }
}

__global__ void _parallel_lower_cube(float * dst, float * src, const struct PMapHelper args){

  // Read arguments
  const int iD = args.sD;
  const int iR = args.sR;
  const int iC = args.sC;
  const int kR = args.kR;
  const int kC = args.kC;
  const int iB = args.sB;
  const int p  = args.padding;
  const int s  = args.stride;
  const int oR = (iR + 2*p - kR) / s + 1;
  const int oC = (iC + 2*p - kC) / s + 1; 
  
  // Get the right loop element
  const int iB_idx   = blockIdx.x;
  const int iD_idx   = blockIdx.y;
  const int oRoC_idx = blockIdx.z;
  const int oR_idx   = oRoC_idx/oC;
  const int oC_idx   = oRoC_idx%oC;
  const int kR_idx   = threadIdx.x;
  const int kC_idx   = threadIdx.y;
  
  const int out_r = iB_idx*oR*oC + oR_idx*oC + oC_idx;
  const int out_c = iD_idx*kR*kC + kR_idx*kC + kC_idx;

  if ( (oR_idx*s-p+kR_idx) >= 0 && (oR_idx*s-p+kR_idx) < iR && (oC_idx*s-p+kC_idx) >= 0 && (oC_idx*s-p+kC_idx) < iC ) {
    dst[out_r*iD*kR*kC + out_c] = src[iB_idx*iC*iR*iD + iD_idx*iR*iC + (oR_idx*s-p+kR_idx)*iC + (oC_idx*s-p+kC_idx)];
  } else {
    dst[out_r*iD*kR*kC + out_c] = 0;
  }
}

__global__ void _parallel_inverse_lower_cube(float * dst, float * src, const struct _inverse_lower_cube_arg_helper args){

  // Read arguments
  const int oC = args.data_output_width;
  const int oR = args.data_output_height;
  const int k = args.kernel_size;
  const int s = args.stride;
  const int p = args.padding;
  const int iR = args.iR;
  const int iC = args.iC;
  const int iD = args.iD;
  const unsigned int iB = args.iB;

  // Get the right loop element
  const int i = blockDim.x * blockIdx.x + threadIdx.x;

  // SHADJIS TODO: These / and % not needed if using multi-dimensional blocks
  const int b =   (i / (iC*iR*iD));
  const int tmp = (i % (iC*iR*iD));
  const int c =  tmp / (iC * iR);
  const int h = (tmp / iC) % iR + p;
  const int w =  tmp % iC + p;
  
  const int w_col_start = (w < k) ? 0 : (w - k) / s + 1;
  const int w_col_end = device_min(w / s + 1, oC);
  const int h_col_start = (h < k) ? 0 : (h - k) / s + 1;
  const int h_col_end = device_min(h / s + 1, oR);
  
  // SHADJIS TODO: Not sure why but the way we store batches is
  // different from Caffe so this part had to be changed. Probably
  // something to do with the old/unnecessary CcT decision to flip
  // the gemm order everywhere. So instead of storing each batch
  // of src one after the other we interleave them. I think this is
  // pretty stupid but I don't feel like rewriting everything now.
  // Probably leave that for when we do fusion.
  const int offset = (c*k*k*iB + h*k*iB + w*iB + b)*oR*oC;
  const int coeff_h_col = (1 - s*k*oR*iB)*oC;
  const int coeff_w_col = (1 - s*oR*oC*iB);
  float sum = 0;
  for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
    for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
      sum += src[offset + h_col * coeff_h_col + w_col * coeff_w_col];
    }
  }
  dst[i] = sum;
}

template<FPMAP_ID f_id, FPMAP_DATA_READC f_data>
__global__ void _spmap_readc(float* dst, float * src, PMapHelper args){
	const size_t block_x = blockIdx.x;
	const size_t block_y = blockIdx.y;

	const size_t nCblock = (args.sC + args.sBC-1)/args.sBC;

	Block2D input_block;
	input_block.r = block_x / nCblock;
	input_block.c = block_x % nCblock;
	input_block.d = block_y % args.sD;
	input_block.b = block_y / args.sD;
	input_block.dr = args.sR;
	input_block.dc = args.sC;

	Block2D output_block;
	f_id(&output_block, &input_block, &args);

	const size_t datar = threadIdx.y + input_block.r * args.sBR;
	const size_t datac = threadIdx.x + input_block.c * args.sBC;

	PointIn2DBlock point;
	point.block = input_block;
    
    const size_t src_idx = args.sR * args.sC * (args.sD * input_block.b + input_block.d) +
            datar * args.sC +
            datac;

    // Check if in bounds
    if (datar < args.sR && datac < args.sC)
    {
        point.data = src[src_idx];
        point.r = datar;
        point.c = datac;
        f_data(dst, &output_block, &point, &args);
    }
}

template<FPMAP_ID f_id, FPMAP_DATA_READC f_data>
void GPUDriver::lower_cube(DeviceMemoryPointer * dst, DeviceMemoryPointer * src, 
    const struct PMapHelper args){
    pmap2d_read_coalesce<f_id, f_data>(dst, src, args);
    // lower_cube_helper(dst, src, args);
}

// SHADJIS TODO: This is a more parallel forward lowering which matches the implementation
// on the CPU. But because it flips the lowering (transposes) also need to change cuBLAS
// flags below.
void GPUDriver::lower_cube_helper(DeviceMemoryPointer * dst, DeviceMemoryPointer * src, 
    const struct PMapHelper args){
    
    const int iD = args.sD;
    const int kr = args.kR;
    const int kc = args.kC;
    const int iB = args.sB;
    const int p  = args.padding;
    const int s  = args.stride;
    const int iR = args.sR;
    const int iC = args.sC;
    const int oR = (iR + 2*p - kr) / s + 1;
    const int oC = (iC + 2*p - kc) / s + 1; 
    
    dim3 numBlocks(iB, iD, oR*oC);
    // SHADJIS TODO: Should fix the number of threads (e.g. 256, 1024) since now
    // warps are under-utilized for small k
    dim3 threadsPerBlock(kr, kc);
    // SHADJIS TODO: Call something like _spmap_readc instead
    // SHADJIS TODO: Add a check here for too many blocks like sapply, or make multi-dimensional like _spmap_readc
    hipGetLastError(); // Reset the error status to success
    _parallel_lower_cube<<<numBlocks, threadsPerBlock>>>((float*) dst->ptr, (float *) src->ptr, args);
    err = hipGetLastError();
    if(err != hipSuccess){
      std::cout << "Fail to launch _parallel_lower_cube"  << "  ERROR " << err << std::endl;
      assert(false);
    }
    hipDeviceSynchronize();
    err = hipGetLastError();
    if(err != hipSuccess){
      std::cout << "Fail to sync _parallel_lower_cube"  << "  ERROR " << err << std::endl;
      assert(false);
    }
}


// Note: lower_cube and also inverse_lower_cube are special-case functions, i.e.
// they do not use parallel map + kernel callbacks. They could use that interface
// but it may be easier for fusion to keep them separate.
void GPUDriver::inverse_lower_cube(DeviceMemoryPointer * dst, DeviceMemoryPointer * src, 
    const struct _inverse_lower_cube_arg_helper args){

    const int iD = args.iD;
    const int iR = args.iR;
    const int iC = args.iC;
    const unsigned int iB = args.iB;
    const int num_parallel_elements = iR*iC*iD*iB;
    int blocksPerGrid = (num_parallel_elements + threadsPerBlock - 1) / threadsPerBlock;
    
    // SHADJIS TODO: Call something like _spmap_readc instead
    // SHADJIS TODO: Add a check here for too many blocks like sapply, or make multi-dimensional like _spmap_readc
    hipGetLastError(); // Reset the error status to success
    _parallel_inverse_lower_cube<<<blocksPerGrid, threadsPerBlock>>>((float*) dst->ptr, (float *) src->ptr, args);
    err = hipGetLastError();
    if(err != hipSuccess){
      std::cout << "Fail to launch _parallel_inverse_lower_cube"  << "  ERROR " << err << std::endl;
      assert(false);
    }
    hipDeviceSynchronize();
    err = hipGetLastError();
    if(err != hipSuccess){
      std::cout << "Fail to sync _parallel_inverse_lower_cube"  << "  ERROR " << err << std::endl;
      assert(false);
    }
}


// SHADJIS TODO: Why is the interface for this is different from parallel_map?
// Here we pass in the args directly whereas parallel_map gets pointers to
// the args already allocated on the device
template<FPMAP_ID f_id, FPMAP_DATA_READC f_data>
void GPUDriver::pmap2d_read_coalesce(DeviceMemoryPointer * dst, DeviceMemoryPointer * src, 
    const struct PMapHelper args){

	// input block sizes
	size_t sBR = args.sBR, sBC = args.sBC;
    
	dim3 threadsPerBlock(sBC, sBR);	// trivial impl -- each input pixel is a single thread
	// The number of blocks and threads are chosen to map to each pixel in input (1 thread/pixel)
	dim3 numBlocks(((args.sR + sBR-1)/sBR)*((args.sC + sBC-1)/sBC), args.sD*args.sB);

	hipGetLastError(); // Reset the error status to success
	_spmap_readc<f_id,f_data><<<numBlocks, threadsPerBlock>>>((float*) dst->ptr, (float*) src->ptr, args);
	err = hipGetLastError();
	if(err != hipSuccess){
	  std::cout << "Fail to launch _spmap_readc"  << "  ERROR " << err << std::endl;
	  assert(false);
	}
	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess){
	  std::cout << "Fail to hipDeviceSynchronize _spmap_readc"  << "  ERROR " << err << std::endl;
	  assert(false);
	}

}


GPUDriver::GPUDriver(){
    hipblasCreate(&handle);
}

DeviceMemoryPointer * GPUDriver::get_device_pointer(void * ptr, size_t size_in_byte){
	// TODO: This has memory leak! Refactor it!
	return new DeviceMemoryPointer_Local_GPURAM(gpu_id, ptr, size_in_byte);
}

void GPUDriver::malloc(DeviceMemoryPointer * dst){
	hipMalloc((void**)&dst->ptr, dst->size_in_byte);
}

void GPUDriver::free(DeviceMemoryPointer * dst){
	hipFree(dst->ptr);
}

void GPUDriver::memcpy(DeviceMemoryPointer * dst, DeviceMemoryPointer * src){
	#ifdef _DO_ASSERT
	assert(dst->size_in_byte == src->size_in_byte);
	#endif
	if(src->type == DEVICEMEMORY_LOCAL_RAM){
  		hipMemcpy(dst->ptr, src->ptr, dst->size_in_byte, hipMemcpyHostToDevice);
	}else if(dst->type == DEVICEMEMORY_LOCAL_RAM){
  		hipMemcpy(dst->ptr, src->ptr, dst->size_in_byte, hipMemcpyDeviceToHost);
	}else{
		hipMemcpy(dst->ptr, src->ptr, dst->size_in_byte, hipMemcpyDeviceToDevice);
	}
}

void GPUDriver::memset(DeviceMemoryPointer * dst, const char value){
	#ifdef _DO_ASSERT
	assert(dst->type==DEVICEMEMORY_LOCAL_RAM);
	#endif
	hipMemset(dst->ptr, value, dst->size_in_byte);
}

template<FUNC_IDX_MAPPING f_dst_pos, FUNC_MM_MAPPING func>
void GPUDriver::parallel_map(DeviceMemoryPointer * dst, DeviceMemoryPointer * src, 
size_t src_skip, DeviceMemoryPointer * const f_dst_pos_curry, DeviceMemoryPointer * const func_curry){

	// create a device version of func_curry
	void * d_func_curry;
	hipMalloc((void**)&d_func_curry, func_curry->size_in_byte);
	hipMemcpy(d_func_curry, func_curry->ptr, func_curry->size_in_byte, hipMemcpyHostToDevice);

	void * d_idx_func_curry;
	hipMalloc((void**)&d_idx_func_curry, f_dst_pos_curry->size_in_byte);
	hipMemcpy(d_idx_func_curry, f_dst_pos_curry->ptr, f_dst_pos_curry->size_in_byte, hipMemcpyHostToDevice);

	// Run.
	hipGetLastError(); // Reset the error status to success
	const int n_elements =  src->size_in_byte / src_skip;
	int blocksPerGrid = (n_elements + threadsPerBlock - 1) / threadsPerBlock;
	// SHADJIS TODO: Why call _spmap and not _spmap_readc?
	_spmap<f_dst_pos,func><<<blocksPerGrid, threadsPerBlock>>>((float*) dst->ptr, (float *) src->ptr,
	  n_elements, src_skip, d_idx_func_curry, d_func_curry);
	err = hipGetLastError();
	if(err != hipSuccess){
	  std::cout << "Fail to launch _spmap"  << "  ERROR " << err << std::endl;
	  assert(false);
	}
	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess){
	  std::cout << "Fail to sync _spmap"  << "  ERROR " << err << std::endl;
	  assert(false);
	}

	hipFree(d_func_curry);
	hipFree(d_idx_func_curry);

}

void GPUDriver::math_saxpy(const float alpha, DeviceMemoryPointer * X, DeviceMemoryPointer * Y) const { 
#ifdef _DO_ASSERT
	assert(X->type==DEVICEMEMORY_LOCAL_RAM);
	assert(Y->type==DEVICEMEMORY_LOCAL_RAM);
	assert(X->size_in_byte==Y->size_in_byte);
#endif
  int n_elements = X->size_in_byte / sizeof(float);
  hipblasStatus_t status = hipblasSaxpy(handle, n_elements, &alpha, (float*)X->ptr, 1, (float*)Y->ptr, 1);
  assert(status == HIPBLAS_STATUS_SUCCESS);
}

void GPUDriver::math_saxpy(const int nElements, const float alpha, float * X, float * Y) const { 
  hipblasStatus_t status = hipblasSaxpy(handle, nElements, &alpha, X, 1, Y, 1);
  assert(status == HIPBLAS_STATUS_SUCCESS);
}

template<FUNC_STRANSFORM func>
void GPUDriver::sapply(DeviceMemoryPointer * dst, DeviceMemoryPointer * const func_curry){
	#ifdef _DO_ASSERT
	assert(dst->type==DEVICEMEMORY_LOCAL_GPURAM);
	assert(dst->size_in_byte % sizeof(float) == 0);
	#endif
	// TODO: Refactoring

	// Second, create a device version of func_curry
	void * d_func_curry;
	hipMalloc((void**)&d_func_curry, func_curry->size_in_byte);
	hipMemcpy(d_func_curry, func_curry->ptr, func_curry->size_in_byte, hipMemcpyHostToDevice);

	hipGetLastError(); // Reset the error status to success
	int n_elements =  dst->size_in_byte / sizeof(float);
	int blocksPerGrid = (n_elements + threadsPerBlock - 1) / threadsPerBlock;

	// Check if we are trying to initialize something huge. In that case call multiple times.
	// SHADJIS TODO: Could make multi-dimensional like _spmap_readc instead
	const int num_calls = (blocksPerGrid + max_cuda_blocks - 1) / max_cuda_blocks;
	if (num_calls > 1) {
		blocksPerGrid = max_cuda_blocks;
	}
	for (int call_counter=0; call_counter < num_calls; ++call_counter)
	{
		_sapply<func><<<blocksPerGrid, threadsPerBlock>>>((float*) (dst->ptr + call_counter*blocksPerGrid*threadsPerBlock), n_elements, d_func_curry);
		err = hipGetLastError();
		if(err != hipSuccess){
			std::cout << "Fail to launch _sapply" << "  ERROR " << err << std::endl;
			assert(false);
		}
		n_elements -= blocksPerGrid*threadsPerBlock; // Decrement #elements left to process
	}
	hipDeviceSynchronize();
	err = hipGetLastError();
	assert(err == hipSuccess);

	hipFree(d_func_curry);
}

void GPUDriver::math_saxpby(const float alpha, DeviceMemoryPointer * X, const float beta, DeviceMemoryPointer * Y) const { 
#ifdef _DO_ASSERT
  assert(X->size_in_byte == Y->size_in_byte);
  assert(X->size_in_byte % sizeof(float) == 0);
#endif

  int n_elements = X->size_in_byte / sizeof(float);
  hipblasStatus_t status = hipblasSscal(handle, n_elements, &beta, (float*)Y->ptr, 1);
  assert(status == HIPBLAS_STATUS_SUCCESS);

  status = hipblasSaxpy(handle, n_elements, &alpha, (float*)X->ptr, 1, (float*)Y->ptr, 1);
  assert(status == HIPBLAS_STATUS_SUCCESS);

}

void GPUDriver::math_saxpby(const int nElements, const float alpha, float * X, const float beta, float * Y) const { 
  hipblasStatus_t status = hipblasSscal(handle, nElements, &beta, Y, 1);
  assert(status == HIPBLAS_STATUS_SUCCESS);

  status = hipblasSaxpy(handle, nElements, &alpha, X, 1, Y, 1);
  assert(status == HIPBLAS_STATUS_SUCCESS);

}

void GPUDriver::set_num_threads(const int nThreads) { 
}


void GPUDriver::sgemm(const enum CBLAS_ORDER order, CBLAS_TRANSPOSE TA, CBLAS_TRANSPOSE TB, 
    int M, int N, int K, float alpha, float * pA, int LDA, float * pB, int LDB,
    float beta, float * pC, int LDC){
  
	// SHADJIS TODO: See comment in Kernel.h regarding transpose. For the CPU it is fastest 
	// to lower like equation 4 of "Formulation of Type 1 Lowering with Padding and Stride"
	// but the GPU currently lowers as the transpose of what the CPU does. For now I change
	// the parameters in here to match. It's pretty complicated to get these cuBLAS parameters
	// right because cuBLAS also assumes things are stored in column-major order. It's made
	// more complicated because the lowering on CPU and GPU differs (by transpose), so making
	// the lowered versions match would make this easier to follow.

	if(TA == CblasNoTrans && TB == CblasNoTrans){

		hipblasOperation_t ta = HIPBLAS_OP_N;
		// tb should also be no trans, but is transposed to match cpu lowering
		hipblasOperation_t tb = HIPBLAS_OP_T; 

		// cublas expects col major, so we change the parameters accordingly
		status = hipblasSgemm(handle, tb, ta, N, M, K, &alpha, 
			pB, K, pA, K, &beta, pC, N); 

		hipDeviceSynchronize();
		err = hipGetLastError();
		assert(err == hipSuccess);

		assert(status == HIPBLAS_STATUS_SUCCESS);

	}else if(TA == CblasTrans && TB == CblasNoTrans){

		hipblasOperation_t ta = HIPBLAS_OP_T;
		hipblasOperation_t tb = HIPBLAS_OP_N;

		// cublas expects col major, so we change the parameters accordingly
		status = hipblasSgemm(handle, tb, ta, N, M, K, &alpha, 
			pB, N, pA, M, &beta, pC, N); 

		hipDeviceSynchronize();
		err = hipGetLastError();
		assert(err == hipSuccess);

		assert(status == HIPBLAS_STATUS_SUCCESS);

	}else if(TA == CblasNoTrans && TB == CblasTrans){

		hipblasOperation_t ta = HIPBLAS_OP_N;
		// tb should be trans, but is transposed to match cpu lowering
		hipblasOperation_t tb = HIPBLAS_OP_N;

		// cublas expects col major, so we change the parameters accordingly
		status = hipblasSgemm(handle, tb, ta, N, M, K, &alpha, 
			pB, N, pA, K, &beta, pC, N); 

		hipDeviceSynchronize();
		err = hipGetLastError();
		assert(err == hipSuccess);

		assert(status == HIPBLAS_STATUS_SUCCESS);

	}else if(TA == CblasTrans && TB == CblasTrans){

		hipblasOperation_t ta = HIPBLAS_OP_T;
		hipblasOperation_t tb = HIPBLAS_OP_T;

		// cublas expects col major, so we change the parameters accordingly
		status = hipblasSgemm(handle, tb, ta, N, M, K, &alpha, 
			pB, K, pA, M, &beta, pC, N); 

		hipDeviceSynchronize();
		err = hipGetLastError();
		assert(err == hipSuccess);

		assert(status == HIPBLAS_STATUS_SUCCESS);

	}else{
		assert(false);
	}

}

template<FUNC_SREDUCE func>
void GPUDriver::selementwise_reduce2(DeviceMemoryPointer * dst, DeviceMemoryPointer * src1, 
DeviceMemoryPointer * src2, DeviceMemoryPointer * const func_curry){ 

	#ifdef _DO_ASSERT
	assert(dst->size_in_byte == src1->size_in_byte);
	assert(dst->size_in_byte == src2->size_in_byte);
	assert(dst->size_in_byte % sizeof(float) == 0);
	#endif

	// create a device version of func_curry
	void * d_func_curry;
	hipMalloc((void**)&d_func_curry, func_curry->size_in_byte);
	hipMemcpy(d_func_curry, func_curry->ptr, func_curry->size_in_byte, hipMemcpyHostToDevice);

	// Run.
	const int n_elements =  dst->size_in_byte / sizeof(float);
	int blocksPerGrid = (n_elements + threadsPerBlock - 1) / threadsPerBlock;
	_sreduce<func><<<blocksPerGrid, threadsPerBlock>>>((float*) dst->ptr, n_elements, 
	  (float*) src1->ptr, (float*) src2->ptr, d_func_curry);
	err = hipGetLastError();
	if(err != hipSuccess){
	  std::cout << "Fail to launch _sreduce" << std::endl;
	  assert(false);
	}
	hipDeviceSynchronize();
	err = hipGetLastError();
	assert(err == hipSuccess);


}

/**
* This function is called only once. So its speed does not matter.
* TODO: Wrap this up with CURAND.
**/
void GPUDriver::sinitialize_xavier(DeviceMemoryPointer *arr, const size_t n_batch) {
	const size_t n_arr_elements = arr->size_in_byte / sizeof(float);
	const size_t fan_in = n_arr_elements / n_batch;
	const float scale = sqrt(3.0 / fan_in);

	mt19937 gen(rd());
	uniform_real_distribution<float> uni(-scale, scale);
	float * temp = new float[n_arr_elements];
	for(int i=0;i<n_arr_elements;i++){
	  temp[i] = uni(gen);
	}
	hipMemcpy(arr->ptr, temp, arr->size_in_byte, hipMemcpyHostToDevice);
	delete[] temp;
	}

/**
* This function is called only once. So its speed does not matter.
* TODO: Wrap this up with CURAND.
**/
void GPUDriver::sbernoulli_initialize(DeviceMemoryPointer *arr, const float p) {
const size_t n_arr_elements = arr->size_in_byte / sizeof(float);

	mt19937 gen(rd());
	bernoulli_distribution bern(p);
	float * temp = new float[n_arr_elements];
	for(int i=0;i<n_arr_elements;i++){
	  temp[i] = bern(gen);
	}
	hipMemcpy(arr->ptr, temp, arr->size_in_byte, hipMemcpyHostToDevice);
	delete[] temp;

}

/**
* This function is called only once. So its speed does not matter.
* TODO: Wrap this up with CURAND.
**/
void GPUDriver::sgaussian_initialize(DeviceMemoryPointer *arr, const float mean, const float std_dev) {
const size_t n_arr_elements = arr->size_in_byte / sizeof(float);
	mt19937 gen(rd());
	normal_distribution<float> gaussian(mean, std_dev);
	float * temp = new float[n_arr_elements];
	for(int i=0;i<n_arr_elements;i++){
	  temp[i] = gaussian(gen);
	}
	hipMemcpy(arr->ptr, temp, arr->size_in_byte, hipMemcpyHostToDevice);
	delete[] temp;

}

void GPUDriver::sconstant_initialize(DeviceMemoryPointer *arr, const float value){
    DeviceMemoryPointer_Local_RAM pvalue((void*)&value, sizeof(float));
    sapply<__sconstant_initialize_helper>(arr, &pvalue);
}

void * GPUDriver::choose_ptr(void * host, void * device){
	return device;
}

/**
 * This is necessary for template to be instantiated.
 */
template void GPUDriver::pmap2d_read_coalesce<_fpmap_id,_fmap_lower>(DeviceMemoryPointer * dst, 
	DeviceMemoryPointer * src, const struct PMapHelper args);

// SHADJIS TODO: No need to template this if we switch to new lowering
template void GPUDriver::lower_cube<_fpmap_id,_fmap_lower>(DeviceMemoryPointer * dst, 
	DeviceMemoryPointer * src, const struct PMapHelper args);

template void GPUDriver::pmap2d_read_coalesce<_fpmap_id,_fmap_remap>(DeviceMemoryPointer * dst, 
	DeviceMemoryPointer * src, const struct PMapHelper args);

/** All template instantiations for parallel_map **/
template void GPUDriver::parallel_map<_f_idx_strid4_copy,_f_strid4_copy>(DeviceMemoryPointer * dst,
    DeviceMemoryPointer * src, size_t src_skip, DeviceMemoryPointer * const f_dst_pos_curry,
    DeviceMemoryPointer * const func_curry);
// inverse_lower_cube
template void GPUDriver::parallel_map<_f_src_to_dst_inverse_lower_cube,_f_inverse_lower_cube>(DeviceMemoryPointer * dst,
    DeviceMemoryPointer * src, size_t src_skip, DeviceMemoryPointer * const f_dst_pos_curry,
    DeviceMemoryPointer * const func_curry);
// Conv/FC Bias forward
template void GPUDriver::parallel_map<_f_src_to_dst_bias_forward,_f_bias_forward>(DeviceMemoryPointer * dst,
    DeviceMemoryPointer * src, size_t src_skip, DeviceMemoryPointer * const f_dst_pos_curry,
    DeviceMemoryPointer * const func_curry);
// Conv/FC Bias backward
template void GPUDriver::parallel_map<_f_src_to_dst_bias_backward,_f_bias_backward>(DeviceMemoryPointer * dst,
    DeviceMemoryPointer * src, size_t src_skip, DeviceMemoryPointer * const f_dst_pos_curry,
    DeviceMemoryPointer * const func_curry);
// ReLU forward
template void GPUDriver::parallel_map<_f_src_to_dst_relu_forward,_f_relu_forward>(DeviceMemoryPointer * dst,
    DeviceMemoryPointer * src, size_t src_skip, DeviceMemoryPointer * const f_dst_pos_curry,
    DeviceMemoryPointer * const func_curry);
// ReLU backward
template void GPUDriver::parallel_map<_f_src_to_dst_relu_backward,_f_relu_backward>(DeviceMemoryPointer * dst,
    DeviceMemoryPointer * src, size_t src_skip, DeviceMemoryPointer * const f_dst_pos_curry,
    DeviceMemoryPointer * const func_curry);
// Dropout forward train
template void GPUDriver::parallel_map<_f_src_to_dst_dropout_forward,_f_dropout_forward_train>(DeviceMemoryPointer * dst,
    DeviceMemoryPointer * src, size_t src_skip, DeviceMemoryPointer * const f_dst_pos_curry,
    DeviceMemoryPointer * const func_curry);
// Dropout forward test
template void GPUDriver::parallel_map<_f_src_to_dst_dropout_forward,_f_dropout_forward_test>(DeviceMemoryPointer * dst,
    DeviceMemoryPointer * src, size_t src_skip, DeviceMemoryPointer * const f_dst_pos_curry,
    DeviceMemoryPointer * const func_curry);
// Pool forward
template void GPUDriver::parallel_map<_f_src_to_dst_pool_forward,_f_pool_forward>(DeviceMemoryPointer * dst,
    DeviceMemoryPointer * src, size_t src_skip, DeviceMemoryPointer * const f_dst_pos_curry,
    DeviceMemoryPointer * const func_curry);
// Pool backward
template void GPUDriver::parallel_map<_f_src_to_dst_pool_backward,_f_pool_backward>(DeviceMemoryPointer * dst,
    DeviceMemoryPointer * src, size_t src_skip, DeviceMemoryPointer * const f_dst_pos_curry,
    DeviceMemoryPointer * const func_curry);
// LRN forward
template void GPUDriver::parallel_map<_f_src_to_dst_lrn_forward,_f_lrn_forward>(DeviceMemoryPointer * dst,
    DeviceMemoryPointer * src, size_t src_skip, DeviceMemoryPointer * const f_dst_pos_curry,
    DeviceMemoryPointer * const func_curry);
// LRN forward normalize
template void GPUDriver::parallel_map<_f_src_to_dst_lrn_forward,_f_lrn_forward_normalize>(DeviceMemoryPointer * dst,
    DeviceMemoryPointer * src, size_t src_skip, DeviceMemoryPointer * const f_dst_pos_curry,
    DeviceMemoryPointer * const func_curry);
// LRN backward
template void GPUDriver::parallel_map<_f_src_to_dst_lrn_backward,_f_lrn_backward>(DeviceMemoryPointer * dst,
    DeviceMemoryPointer * src, size_t src_skip, DeviceMemoryPointer * const f_dst_pos_curry,
    DeviceMemoryPointer * const func_curry);
// Softmax forward
template void GPUDriver::parallel_map<_f_src_to_dst_softmax_forward,_f_softmax_forward>(DeviceMemoryPointer * dst,
    DeviceMemoryPointer * src, size_t src_skip, DeviceMemoryPointer * const f_dst_pos_curry,
    DeviceMemoryPointer * const func_curry);
// Softmax backward
template void GPUDriver::parallel_map<_f_src_to_dst_softmax_backward,_f_softmax_backward>(DeviceMemoryPointer * dst,
    DeviceMemoryPointer * src, size_t src_skip, DeviceMemoryPointer * const f_dst_pos_curry,
    DeviceMemoryPointer * const func_curry);

template void GPUDriver::sapply<_f_add_one>(DeviceMemoryPointer * dst, DeviceMemoryPointer * const func_curry);

template void GPUDriver::sapply<_f_set>(DeviceMemoryPointer * dst, DeviceMemoryPointer * const func_curry);

template void GPUDriver::selementwise_reduce2<_f_reduce>(DeviceMemoryPointer * dst, 
	DeviceMemoryPointer * src1, DeviceMemoryPointer * src2, DeviceMemoryPointer * const func_curry);

template void GPUDriver::selementwise_reduce2<_f_reduce_mul>(DeviceMemoryPointer * dst, 
	DeviceMemoryPointer * src1, DeviceMemoryPointer * src2, DeviceMemoryPointer * const func_curry);

template void GPUDriver::selementwise_reduce2<_f_reduce_tanhgrad>(DeviceMemoryPointer * dst, 
	DeviceMemoryPointer * src1, DeviceMemoryPointer * src2, DeviceMemoryPointer * const func_curry);

