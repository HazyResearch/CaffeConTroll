#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "src/kernels/lowering.hxx"

#include "DeviceDriver.h"
#include "DeviceDriver_GPU.h"

__host__ __device__ float __sconstant_initialize_helper(float a, void * arg){
  return *((float*)arg);
}
__device__ FUNC_STRANSFORM _sconstant_initialize_helper = __sconstant_initialize_helper;

__global__ void _sapply(float * dst, int numElements, FUNC_STRANSFORM func, 
            void * const func_curry){
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i < numElements){
    dst[i] = (*func)(dst[i], func_curry);
  }
}


__global__ void _sreduce(float * dst, int numElements, float * src1, float * src2,
            FUNC_SREDUCE func, void * const func_curry){
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i < numElements){
    dst[i] = (*func)(src1[i], src2[i], func_curry);
  }
}


__global__ void _spmap(float * dst, float * src, int numElements, int srcSkip,
  FUNC_IDX_MAPPING idx_func, void * const idx_func_curry,
  FUNC_MM_MAPPING func, void * const func_curry){

  int i = blockDim.x * blockIdx.x + threadIdx.x;
  i = i * srcSkip;
  int src_idx, dst_idx;

  for(int j=0; j<srcSkip; j++){
    src_idx = i + j;
    if(src_idx < numElements){
      dst_idx = (*idx_func)(src_idx, idx_func_curry);
      (*func)(&dst[dst_idx], &src[src_idx], func_curry);
    }
  }
}


__device__ 
void _fpmap_id2(Block2D * const output_block, const Block2D * const input_block, const PMapHelper * const args){
	output_block->r = 0;
	output_block->c = 0;
	output_block->d = 0;
	output_block->d = 0;
	output_block->dr = args->kR;
	output_block->dc = args->kC;
}

__device__ 
void _fmap_lower2(float * output, const Block2D * const output_block, const PointIn2DBlock * const input_point, const PMapHelper * const args){
	
	const size_t ir = input_point->r;
	const size_t ic = input_point->c;
	const size_t ib = input_point->block.b;
	const size_t id = input_point->block.d;

	const size_t kR = args->kR;
	const size_t kC = args->kC;
	const size_t iR = args->sR;
	const size_t iC = args->sC;
	const size_t iB = args->sB;
	const size_t o_base_col = ib * (iR-kR+1)*(iC-kC+1);
	const size_t o_base_row = id * kR * kC;
	const size_t oC = iB * (iR-kR+1)*(iC-kC+1);

	const float input = input_point->data;

	for(int r=ir-kR;r<=ir;r++){
		int dr = ir-r;
		for(int c=ic-kC;c<=ic;c++){
			int dc = ic-c;
			int ocol = r*iC+c;
			int orow = dr*kC+dc;
			int ocol2 = ocol + o_base_col;
			int orow2 = orow + o_base_row;
			// then write to ocol, orow
			if(ocol >= 0 && ocol < (iR-kR+1)*(iC-kC+1) && orow >= 0 && orow < kR*kC){
				output[ocol + orow*oC] = input;
			}
		}
	}
}

template<FPMAP_ID f_id, FPMAP_DATA_READC f_data>
__global__ void _spmap_readc(float* dst, float * src, PMapHelper args){
	const size_t block_x = blockIdx.x;
	const size_t block_y = blockIdx.y;

	const size_t nRblock = args.sR/args.sBR;
	const size_t nCblock = args.sC/args.sBC;

	Block2D input_block;
	input_block.r = block_x / nCblock;
	input_block.c = block_x % nCblock;
	input_block.d = block_y % args.sD;
	input_block.b = block_y / args.sD;
	input_block.dr = args.sR;
	input_block.dc = args.sC;

	Block2D output_block;
	//(*f_id)(&output_block, &input_block, &args);
	//_fpmap_id2(&output_block, &input_block, &args);
	f_id(&output_block, &input_block, &args);

	const size_t datar = threadIdx.y + input_block.r * args.sR;
	const size_t datac = threadIdx.x + input_block.c * args.sC;

	PointIn2DBlock point;
	point.block = input_block;
	point.data = src[
		args.sR * args.sC * (args.sD * input_block.b + input_block.d) +
		datar + args.sC +
		datac
	];
	point.r = threadIdx.y;
	point.c = threadIdx.x;

	//(*f_data)(dst, &output_block, &point, &args);
	//_fmap_lower2(dst, &output_block, &point, &args);
	f_data(dst, &output_block, &point, &args);

}


template<__device__ FPMAP_ID f_id, __device__ FPMAP_DATA_READC f_data>
void GPUDriver::pmap2d_read_coalesce(DeviceMemoryPointer * dst, DeviceMemoryPointer * src, 
    const struct PMapHelper args){

	// First, create host version of func
	//FPMAP_DATA_READC h_func;
	//hipMemcpyFromSymbol(&h_func, HIP_SYMBOL(*f_data), sizeof(FPMAP_DATA_READC));
	//FPMAP_DATA_READC d_myfunc = h_func;

	//FPMAP_ID h_idx_func;
	//hipMemcpyFromSymbol(&h_idx_func, HIP_SYMBOL(*f_id), sizeof(FPMAP_ID));
	//FPMAP_ID d_idx_myfunc = h_idx_func;

	// input block sizes
	size_t sBR = args.sBR, sBC = args.sBC;
	dim3 threadsPerBlock(sBC, sBR);	// trivial impl -- each input pixel is a single thread
	dim3 numBlocks(args.sR*args.sC/sBC/sBR, args.sD*args.sB);

	//_spmap_readc<_fpmap_id,_fmap_lower><<<numBlocks, threadsPerBlock>>>((float*) dst->ptr, (float*) src->ptr, args);

	_spmap_readc<f_id,f_data><<<numBlocks, threadsPerBlock>>>((float*) dst->ptr, (float*) src->ptr, args);
	err = hipGetLastError();
	if(err != hipSuccess){
	  std::cout << "Fail to launch _spmap_readc"  << "  ERROR " << err << std::endl;
	  assert(false);
	}
	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess){
	  std::cout << "Fail to hipDeviceSynchronize _spmap_readc"  << "  ERROR " << err << std::endl;
	  assert(false);
	}

}


GPUDriver::GPUDriver(){
    hipblasCreate(&handle);
}

DeviceMemoryPointer * GPUDriver::get_device_pointer(void * ptr, size_t size_in_byte){
	// TODO: This has memory leak! Refactor it!
	return new DeviceMemoryPointer_Local_GPURAM(gpu_id, ptr, size_in_byte);
}

void GPUDriver::malloc(DeviceMemoryPointer * dst){
	hipMalloc((void**)&dst->ptr, dst->size_in_byte);
}

void GPUDriver::free(DeviceMemoryPointer * dst){
	hipFree(dst->ptr);
}

void GPUDriver::memcpy(DeviceMemoryPointer * dst, DeviceMemoryPointer * src){
	#ifdef _DO_ASSERT
	assert(dst->type==DEVICEMEMORY_LOCAL_RAM);
	assert(src->type==DEVICEMEMORY_LOCAL_RAM);
	assert(dst->size_in_byte == src->size_in_byte);
	#endif
	hipMemcpy(dst->ptr, src->ptr, dst->size_in_byte, hipMemcpyDeviceToDevice);
}

void GPUDriver::memset(DeviceMemoryPointer * dst, const char value){
	#ifdef _DO_ASSERT
	assert(dst->type==DEVICEMEMORY_LOCAL_RAM);
	#endif
	hipMemset(dst->ptr, value, dst->size_in_byte);
}

void GPUDriver::parallel_map(DeviceMemoryPointer * dst, DeviceMemoryPointer * src, 
size_t src_skip, FUNC_IDX_MAPPING * f_dst_pos, DeviceMemoryPointer * const f_dst_pos_curry,
FUNC_MM_MAPPING * func, DeviceMemoryPointer * const func_curry){

	// First, create host version of func
	FUNC_MM_MAPPING h_func;
	hipMemcpyFromSymbol(&h_func, HIP_SYMBOL(*func), sizeof(FUNC_MM_MAPPING));
	FUNC_MM_MAPPING d_myfunc = h_func;

	FUNC_IDX_MAPPING h_idx_func;
	hipMemcpyFromSymbol(&h_idx_func, HIP_SYMBOL(*f_dst_pos), sizeof(FUNC_IDX_MAPPING));
	FUNC_IDX_MAPPING d_idx_myfunc = h_idx_func;

	// Second, create a device version of func_curry
	void * d_func_curry;
	hipMalloc((void**)&d_func_curry, func_curry->size_in_byte);
	hipMemcpy(d_func_curry, func_curry->ptr, func_curry->size_in_byte, hipMemcpyHostToDevice);

	void * d_idx_func_curry;
	hipMalloc((void**)&d_idx_func_curry, f_dst_pos_curry->size_in_byte);
	hipMemcpy(d_idx_func_curry, f_dst_pos_curry->ptr, f_dst_pos_curry->size_in_byte, hipMemcpyHostToDevice);

	// Run.
	const int n_elements =  dst->size_in_byte / sizeof(float);
	int blocksPerGrid = (n_elements/src_skip + 1 + threadsPerBlock - 1) / threadsPerBlock;
	_spmap<<<blocksPerGrid, threadsPerBlock>>>((float*) dst->ptr, (float *) src->ptr,
	  n_elements, src_skip, d_idx_myfunc, d_idx_func_curry, d_myfunc, d_func_curry);
	err = hipGetLastError();
	if(err != hipSuccess){
	  std::cout << "Fail to launch _spmap"  << "  ERROR " << err << std::endl;
	  assert(false);
	}
	hipDeviceSynchronize();
	err = hipGetLastError();
	assert(err == hipSuccess);

	hipFree(d_func_curry);
	hipFree(d_idx_func_curry);

}

void GPUDriver::smath_axpy(const float alpha, DeviceMemoryPointer * X, DeviceMemoryPointer * Y)  { 
#ifdef _DO_ASSERT
	assert(X->type==DEVICEMEMORY_LOCAL_RAM);
	assert(Y->type==DEVICEMEMORY_LOCAL_RAM);
	assert(X->size_in_byte==Y->size_in_byte);
#endif
  int n_elements = X->size_in_byte / sizeof(float);
  status = hipblasSaxpy(handle, n_elements, &alpha, (float*)X->ptr, 1, (float*)Y->ptr, 1);
  assert(status == HIPBLAS_STATUS_SUCCESS);
}

void GPUDriver::sapply(DeviceMemoryPointer * dst, FUNC_STRANSFORM * func, DeviceMemoryPointer * const func_curry){
	#ifdef _DO_ASSERT
	assert(dst->type==DEVICEMEMORY_LOCAL_RAM);
	assert(dst->size_in_byte % sizeof(float) == 0);
	#endif
	// TODO: Refactoring

	// First, create host version of func
	FUNC_STRANSFORM h_func;
	hipMemcpyFromSymbol(&h_func, HIP_SYMBOL(*func), sizeof(FUNC_STRANSFORM));
	FUNC_STRANSFORM d_myfunc = h_func;

	// Second, create a device version of func_curry
	void * d_func_curry;
	hipMalloc((void**)&d_func_curry, func_curry->size_in_byte);
	hipMemcpy(d_func_curry, func_curry->ptr, func_curry->size_in_byte, hipMemcpyHostToDevice);

	// Run.
	const int n_elements =  dst->size_in_byte / sizeof(float);
	int blocksPerGrid = (n_elements + threadsPerBlock - 1) / threadsPerBlock;
	_sapply<<<blocksPerGrid, threadsPerBlock>>>((float*) dst->ptr, n_elements, d_myfunc, d_func_curry);
	err = hipGetLastError();
	if(err != hipSuccess){
	  std::cout << "Fail to launch _sapply" << "  ERROR " << err << std::endl;
	  assert(false);
	}
	hipDeviceSynchronize();
	err = hipGetLastError();
	assert(err == hipSuccess);

	hipFree(d_func_curry);
}

void GPUDriver::smath_axpby(const float alpha, DeviceMemoryPointer * X, const float beta, DeviceMemoryPointer * Y) { 
#ifdef _DO_ASSERT
  assert(X->size_in_byte == Y->size_in_byte);
  assert(X->size_in_byte % sizeof(float) == 0);
#endif

  int n_elements = X->size_in_byte / sizeof(float);
  status = hipblasSscal(handle, n_elements, &beta, (float*)Y->ptr, 1);
  assert(status == HIPBLAS_STATUS_SUCCESS);

  status = hipblasSaxpy(handle, n_elements, &alpha, (float*)X->ptr, 1, (float*)Y->ptr, 1);
  assert(status == HIPBLAS_STATUS_SUCCESS);

}

void GPUDriver::set_num_threads(const int nThreads) { 
}


void GPUDriver::sgemm(const enum CBLAS_ORDER order, CBLAS_TRANSPOSE TA, CBLAS_TRANSPOSE TB, 
    int M, int N, int K, float alpha, float * pA, int LDA, float * pB, int LDB,
    float beta, float * pC, int LDC){
  //cblas_sgemm(order, TA, TB, M, N, K, alpha,
  //  pA, LDA,
  //  pB, LDB,
  //  beta, pC, LDC);
}

void GPUDriver::selementwise_reduce2(DeviceMemoryPointer * dst, DeviceMemoryPointer * src1, 
DeviceMemoryPointer * src2, FUNC_SREDUCE * func, DeviceMemoryPointer * const func_curry){ 

	#ifdef _DO_ASSERT
	assert(dst->size_in_byte == src1->size_in_byte);
	assert(dst->size_in_byte == src2->size_in_byte);
	assert(dst->size_in_byte % sizeof(float) == 0);
	#endif
	// First, create host version of func
	FUNC_SREDUCE h_func;
	hipMemcpyFromSymbol(&h_func, HIP_SYMBOL(*func), sizeof(FUNC_SREDUCE));
	FUNC_SREDUCE d_myfunc = h_func;

	// Second, create a device version of func_curry
	void * d_func_curry;
	hipMalloc((void**)&d_func_curry, func_curry->size_in_byte);
	hipMemcpy(d_func_curry, func_curry->ptr, func_curry->size_in_byte, hipMemcpyHostToDevice);

	// Run.
	const int n_elements =  dst->size_in_byte / sizeof(float);
	int blocksPerGrid = (n_elements + threadsPerBlock - 1) / threadsPerBlock;
	_sreduce<<<blocksPerGrid, threadsPerBlock>>>((float*) dst->ptr, n_elements, 
	  (float*) src1->ptr, (float*) src2->ptr, d_myfunc, d_func_curry);
	err = hipGetLastError();
	if(err != hipSuccess){
	  std::cout << "Fail to launch _sreduce" << std::endl;
	  assert(false);
	}
	hipDeviceSynchronize();
	err = hipGetLastError();
	assert(err == hipSuccess);


}

FUNC_STRANSFORM * GPUDriver::srand_uni(float lower, float upper, DeviceMemoryPointer * arg){return NULL;}

FUNC_STRANSFORM * GPUDriver::srand_bern(float p, DeviceMemoryPointer * arg){return NULL;}

FUNC_STRANSFORM * GPUDriver::srand_gaussian(float mean, float std_dev, DeviceMemoryPointer * arg){return NULL;}

/**
* This function is called only once. So its speed does not matter.
* TODO: Wrap this up with CURAND.
**/
void GPUDriver::sinitialize_xavier(DeviceMemoryPointer *arr, const size_t n_batch) {
	const size_t n_arr_elements = arr->size_in_byte / sizeof(float);
	const size_t fan_in = n_arr_elements / n_batch;
	const float scale = sqrt(3.0 / fan_in);

	mt19937 gen(rd());
	uniform_real_distribution<float> uni(-scale, scale);
	float * temp = new float[n_arr_elements];
	for(int i=0;i<n_arr_elements;i++){
	  temp[i] = uni(gen);
	}
	hipMemcpy(arr->ptr, temp, arr->size_in_byte, hipMemcpyHostToDevice);
	delete[] temp;
	}

/**
* This function is called only once. So its speed does not matter.
* TODO: Wrap this up with CURAND.
**/
void GPUDriver::sbernoulli_initialize(DeviceMemoryPointer *arr, const float p) {
const size_t n_arr_elements = arr->size_in_byte / sizeof(float);

	mt19937 gen(rd());
	bernoulli_distribution bern(p);
	float * temp = new float[n_arr_elements];
	for(int i=0;i<n_arr_elements;i++){
	  temp[i] = bern(gen);
	}
	hipMemcpy(arr->ptr, temp, arr->size_in_byte, hipMemcpyHostToDevice);
	delete[] temp;

}

/**
* This function is called only once. So its speed does not matter.
* TODO: Wrap this up with CURAND.
**/
void GPUDriver::sgaussian_initialize(DeviceMemoryPointer *arr, const float mean, const float std_dev) {
const size_t n_arr_elements = arr->size_in_byte / sizeof(float);
	mt19937 gen(rd());
	normal_distribution<float> gaussian(mean, std_dev);
	float * temp = new float[n_arr_elements];
	for(int i=0;i<n_arr_elements;i++){
	  temp[i] = gaussian(gen);
	}
	hipMemcpy(arr->ptr, temp, arr->size_in_byte, hipMemcpyHostToDevice);
	delete[] temp;

}

void GPUDriver::sconstant_initialize(DeviceMemoryPointer *arr, const float value){
    DeviceMemoryPointer_Local_RAM pvalue((void*)&value, sizeof(float));
    sapply(arr, 
      (FUNC_STRANSFORM*)this->choose_ptr((void*)&__sconstant_initialize_helper,
                                          (void*)&_sconstant_initialize_helper),
      &pvalue);
}

void * GPUDriver::choose_ptr(void * host, void * device){
	return device;
}

template void GPUDriver::pmap2d_read_coalesce<_fpmap_id,_fmap_lower>(DeviceMemoryPointer * dst, 
	DeviceMemoryPointer * src, const struct PMapHelper args);


